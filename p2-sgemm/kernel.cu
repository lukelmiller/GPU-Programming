/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
#include <math.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/
    // INSERT KERNEL CODE HERE

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float cVal = 0;
    if(row<m && col<n){
        for(int i = 0; i < k; ++i){
            cVal += (A[row*k+i] * B[i*n+col]);
            
        }
        C[row*n+col] = cVal;
    }

}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks

    //INSERT CODE HERE
    int mDim = ceil(m/BLOCK_SIZE)+1;
    int nDim = ceil(n/BLOCK_SIZE)+1;
    dim3 gridDim(nDim, mDim, 1);
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE, 1);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<gridDim, blockDim>>>(m, n, k, A, B, C);


}


