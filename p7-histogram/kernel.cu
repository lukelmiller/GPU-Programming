
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to



// INSERT KERNEL(S) HERE
#define BLOCK_SIZE 512

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

  extern __shared__ unsigned int pBins[];

  for (int i = threadIdx.x; i < num_bins; i+= BLOCK_SIZE){
    pBins[i] = 0;
  }
  __syncthreads();

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while (index < num_elements) {
    atomicAdd(&(pBins[input[index]]), 1);
    index += stride;
  }
  __syncthreads();

  for (int i = threadIdx.x; i < num_bins; i+= BLOCK_SIZE){
    atomicAdd(&(bins[i]), pBins[i]);
  }
  
}

__global__ void convert_kernel(unsigned int *bins32, uint8_t *bins8,
                               unsigned int num_bins) {
  int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  bins8[index] = (bins32[index] > 255) ? 255 : bins32[index];
}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int *input, uint8_t *bins, unsigned int num_elements,
               unsigned int num_bins) {

  // Create 32 bit bins
  unsigned int *bins32;
  hipMalloc((void **)&bins32, num_bins * sizeof(unsigned int));
  hipMemset(bins32, 0, num_bins * sizeof(unsigned int));

  // Launch histogram kernel using 32-bit bins
  dim3 dim_grid, dim_block;
  dim_block.x = 512;
  dim_block.y = dim_block.z = 1;
  dim_grid.x = 30;
  dim_grid.y = dim_grid.z = 1;
  histogram_kernel<<<dim_grid, dim_block, num_bins * sizeof(unsigned int)>>>(
      input, bins32, num_elements, num_bins);

  // Convert 32-bit bins into 8-bit bins
  dim_block.x = 512;
  dim_grid.x = (num_bins - 1) / dim_block.x + 1;
  convert_kernel<<<dim_grid, dim_block>>>(bins32, bins, num_bins);

  // Free allocated device memory
  hipFree(bins32);
}
