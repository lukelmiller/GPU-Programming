/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"
#include <time.h>

int main(int argc, char* argv[])
{
    Timer timer;
    time_t t;
    
    /* Intializes random number generator */
    srand((unsigned) time(&t));    
    
    

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

	Matrix M_h, N_h, P_h; // M: filter, N: input image, P: output image
	Matrix N_d, P_d;
	unsigned imageHeight, imageWidth;
	hipError_t cuda_ret;
	

	/* Read image dimensions */
    if (argc == 1) {
        imageHeight = 600;
        imageWidth = 1000;
    } else if (argc == 2) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[1]);
    } else if (argc == 3) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[2]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./convolution          # Image is 600 x 1000"
           "\n    Usage: ./convolution <m>      # Image is m x m"
           "\n    Usage: ./convolution <m> <n>  # Image is m x n"
           "\n");
        exit(0);
    }

	/* Allocate host memory */
	M_h = allocateMatrix(FILTER_SIZE, FILTER_SIZE);
	N_h = allocateMatrix(imageHeight, imageWidth);
	P_h = allocateMatrix(imageHeight, imageWidth);

	/* Initialize filter and images */
	initMatrix(M_h);
	initMatrix(N_h);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Image: %u x %u\n", imageHeight, imageWidth);
    printf("    Mask: %u x %u\n", FILTER_SIZE, FILTER_SIZE);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

	N_d = allocateDeviceMatrix(imageHeight, imageWidth);
	P_d = allocateDeviceMatrix(imageHeight, imageWidth);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

	/* Copy image to device global memory */
	copyToDeviceMatrix(N_d, N_h);

	/* Copy mask to device constant memory */
    //INSERT CODE HERE

    cuda_ret = hipMemcpyToSymbol(HIP_SYMBOL(M_c), M_h.elements, (FILTER_SIZE * FILTER_SIZE * sizeof(float)));
    if(cuda_ret != hipSuccess) FATAL("Unable to copy make to device constant memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    int h_blocks = imageHeight/TILE_SIZE + 1;
    int w_blocks = imageWidth/TILE_SIZE + 1;

    dim3 dim_grid(w_blocks,h_blocks);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);

	convolution<<<dim_grid, dim_block>>>(N_d, P_d);

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    copyFromDeviceMatrix(P_h, P_d);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(M_h, N_h, P_h);

    // Free memory ------------------------------------------------------------

	freeMatrix(M_h);
	freeMatrix(N_h);
	freeMatrix(P_h);
	freeDeviceMatrix(N_d);
	freeDeviceMatrix(P_d);

	return 0;
}

